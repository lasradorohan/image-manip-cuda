#include "hip/hip_runtime.h"
#include "kernels.h"
#include "cmath"
#include <iostream>

ImageCommand::~ImageCommand() {};

__global__ void blackWhite(uchar4* image, size_t height, size_t width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;
		unsigned char val = 0.299 * image[idx].x + 0.587 * image[idx].y + 0.114 * image[idx].z;
		image[idx].x = val;
		image[idx].y = val;
		image[idx].z = val;
	}
}

void BlackWhiteImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	std::cout << "Executing BlackWhiteImageCommand(height=" << *height << ", width=" << *width << ")\n";
	size_t in_h = *height;
	size_t in_w = *width;
	uchar4* d_image;
	hipMalloc(&d_image, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_image, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	blackWhite <<<dim3(1 + ((in_h - 1) / 32), 1 + ((in_w - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, in_h, in_w);
	hipMemcpy(*image, d_image, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
	//image memory, height and width dont change
}


__global__ void rotate(uchar4* image_in, uchar4* image_out,
	size_t in_h, size_t in_w, size_t out_h, size_t out_w, float phi) {
	int out_x = blockIdx.x * blockDim.x + threadIdx.x;
	int out_y = blockIdx.y * blockDim.y + threadIdx.y;
	int in_x = (out_x - out_w / 2.f) * __cosf(phi) - (out_y - out_h / 2.f) * __sinf(phi) + in_w/2.f;
	int in_y = (out_y - out_h / 2.f) * __cosf(phi) + (out_x - out_w / 2.f) * __sinf(phi) + in_h/2.f;
	if (out_x < out_w && out_y < out_h) {
		if (0 <= in_x && in_x < in_w && 0 <= in_y && in_y < in_h) {
			image_out[out_w * out_y + out_x] = image_in[in_y * in_w + in_x];
		}
		/*else {
			image_out[out_w * out_y + out_x] = {0, 0, 0, 0};
		}*/
	}
}

RotateImageCommand::RotateImageCommand(float phi) : phi(phi) {}

void RotateImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	std::cout << "Executing RotateImageCommand(height=" << *height << ", width=" << *width << ")\n";
	size_t in_h = *height;
	size_t in_w = *width;
	size_t out_w = in_w * cos(phi) + in_h * sin(phi);
	size_t out_h = in_w * sin(phi) + in_h * cos(phi);
	uchar4* image_out = new uchar4[out_w * out_h];
	uchar4* d_in, * d_out;
	hipMalloc(&d_in, in_h * in_w * sizeof(uchar4));
	hipMalloc(&d_out, out_h * out_w * sizeof(uchar4));
	hipMemcpy(d_in, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	rotate <<< dim3(1 + ((out_w - 1) / 32), 1 + ((out_h - 1) / 32), 1), dim3(32, 32, 1) >>> (d_in, d_out, in_h, in_w, out_h, out_w, phi);
	hipMemcpy(image_out, d_out, out_h * out_w * sizeof(uchar4), hipMemcpyDeviceToHost);
	delete[] * image;
	*image = image_out;
	*height = out_h;
	*width = out_w;
}

__global__ void gammaCorrection(uchar4* image, size_t height, size_t width, float gc) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;
		float r, g, b;
		r = image[idx].x;
		g = image[idx].y;
		b = image[idx].z;
		r = 255 * __powf((r / 255), (1 / gc));
		g = 255 * __powf((g / 255), (1 / gc));
		b = 255 * __powf((b / 255), (1 / gc));
		image[idx].x = r;
		image[idx].y = g;
		image[idx].z = b;
	}
}

GammaCorrectionImageCommand::GammaCorrectionImageCommand(float gc) : gc(gc) {}

void GammaCorrectionImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	uchar4* d_image;
	size_t in_h = *height;
	size_t in_w = *width;
	hipMalloc(&d_image, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_image, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	gammaCorrection <<< dim3(1 + ((in_h - 1) / 32), 1 + ((in_w - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, in_h, in_w, gc);
	hipMemcpy(*image, d_image, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
}


__global__ void radial(uchar4* image_in, uchar4* image_out,
	size_t height, size_t width, float k1, float s) {
	int out_x = blockIdx.x * blockDim.x + threadIdx.x;
	int out_y = blockIdx.y * blockDim.y + threadIdx.y;
	float dx = out_x / (float)width - 0.5f;
	float dy = out_y / (float)height - 0.5f;
	float denom = s + k1 * (dx*dx + dy*dy);
	int in_x = (0.5f + dx / denom)*width;
	int in_y = (0.5f + dy / denom)*height;
	if (out_x < width && out_y < height) {
		if (0 <= in_x && in_x < width && 0 <= in_y && in_y < height) {
			image_out[width * out_y + out_x] = image_in[in_y * width + in_x];
		}
		/*else {
			image_out[width * out_y + out_x] = {0, 0, 0, 0};
		}*/
	}
}

RadialDistortionImageCommand::RadialDistortionImageCommand(float k1) : k1(k1) {}

void RadialDistortionImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	size_t in_h = *height;
	size_t in_w = *width;
	uchar4* d_in, * d_out;
	float scale;
	if (k1 > 0) scale = 1 - k1 / 2;
	else scale = 1 - k1 / 4;
	hipMalloc(&d_in, in_h * in_w * sizeof(uchar4));
	hipMalloc(&d_out, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_in, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	radial <<< dim3(1 + ((in_w - 1) / 32), 1 + ((in_h - 1) / 32), 1), dim3(32, 32, 1) >>> (d_in, d_out, in_h, in_w, k1, scale);
	hipMemcpy(*image, d_out, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
}