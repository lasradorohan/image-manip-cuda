#include "hip/hip_runtime.h"
#include "ImageCommands.h"

ImageCommand::~ImageCommand() {};


__global__ void blackWhite(uchar4* image, size_t height, size_t width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;
		unsigned char val = 0.299 * image[idx].x + 0.587 * image[idx].y + 0.114 * image[idx].z;
		image[idx].x = val;
		image[idx].y = val;
		image[idx].z = val;
	}
}

void BlackWhiteImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	size_t in_h = *height;
	size_t in_w = *width;
	uchar4* d_image;
	hipMalloc(&d_image, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_image, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	blackWhite <<<dim3(1 + ((in_h - 1) / 32), 1 + ((in_w - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, in_h, in_w);
	hipMemcpy(*image, d_image, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
	//image memory, height and width dont change
}

std::string BlackWhiteImageCommand::toString() {
	return "BlackWhite()";
}

__global__ void rotate(uchar4* image_in, uchar4* image_out, size_t in_h, size_t in_w, size_t out_h, size_t out_w, float phi) {
	int out_x = blockIdx.x * blockDim.x + threadIdx.x;
	int out_y = blockIdx.y * blockDim.y + threadIdx.y;
	int in_x = (out_x - out_w / 2.f) * __cosf(phi) - (out_y - out_h / 2.f) * __sinf(phi) + in_w/2.f;
	int in_y = (out_y - out_h / 2.f) * __cosf(phi) + (out_x - out_w / 2.f) * __sinf(phi) + in_h/2.f;
	if (out_x < out_w && out_y < out_h) {
		if (0 <= in_x && in_x < in_w && 0 <= in_y && in_y < in_h) {
			image_out[out_w * out_y + out_x] = image_in[in_y * in_w + in_x];
		}
		else {
			image_out[out_w * out_y + out_x] = {0, 0, 0, 0};
		}
	}
}

RotateImageCommand::RotateImageCommand(float phi) : phi(phi * PI / 180.0f) {}

void RotateImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	size_t in_h = *height;
	size_t in_w = *width;
	size_t out_w = in_w * cos(phi) + in_h * sin(phi);
	size_t out_h = in_w * sin(phi) + in_h * cos(phi);
	uchar4* image_out = new uchar4[out_w * out_h];
	uchar4* d_in, * d_out;
	hipMalloc(&d_in, in_h * in_w * sizeof(uchar4));
	hipMalloc(&d_out, out_h * out_w * sizeof(uchar4));
	hipMemcpy(d_in, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	rotate <<< dim3(1 + ((out_w - 1) / 32), 1 + ((out_h - 1) / 32), 1), dim3(32, 32, 1) >>> (d_in, d_out, in_h, in_w, out_h, out_w, phi);
	hipMemcpy(image_out, d_out, out_h * out_w * sizeof(uchar4), hipMemcpyDeviceToHost);
	delete[] * image;
	*image = image_out;
	*height = out_h;
	*width = out_w;
}

std::string RotateImageCommand::toString() {
	return "Rotate(" + std::to_string(phi) + ")";
}

__global__ void gammaCorrection(uchar4* image, size_t height, size_t width, float gc) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;
		float r, g, b;
		r = image[idx].x;
		g = image[idx].y;
		b = image[idx].z;
		r = 255 * __powf((r / 255), (1 / gc));
		g = 255 * __powf((g / 255), (1 / gc));
		b = 255 * __powf((b / 255), (1 / gc));
		image[idx].x = r;
		image[idx].y = g;
		image[idx].z = b;
	}
}

GammaCorrectionImageCommand::GammaCorrectionImageCommand(float gc) : gc(gc) {}

void GammaCorrectionImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	uchar4* d_image;
	size_t in_h = *height;
	size_t in_w = *width;
	hipMalloc(&d_image, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_image, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	gammaCorrection <<< dim3(1 + ((in_h - 1) / 32), 1 + ((in_w - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, in_h, in_w, gc);
	hipMemcpy(*image, d_image, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
}

std::string GammaCorrectionImageCommand::toString() {
	return "GammaCorrection(" + std::to_string(gc) + ")";
}

__global__ void radial(uchar4* image_in, uchar4* image_out, size_t height, size_t width, float k1, float s) {
	int out_x = blockIdx.x * blockDim.x + threadIdx.x;
	int out_y = blockIdx.y * blockDim.y + threadIdx.y;
	float dx = out_x / (float)width - 0.5f;
	float dy = out_y / (float)height - 0.5f;
	float denom = s + k1 * (dx*dx + dy*dy);
	int in_x = (0.5f + dx / denom)*width;
	int in_y = (0.5f + dy / denom)*height;
	if (out_x < width && out_y < height) {
		if (0 <= in_x && in_x < width && 0 <= in_y && in_y < height) {
			image_out[width * out_y + out_x] = image_in[in_y * width + in_x];
		}
		/*else {
			image_out[width * out_y + out_x] = {0, 0, 0, 0};
		}*/
	}
}

RadialDistortionImageCommand::RadialDistortionImageCommand(float k1) : k1(k1) {}

void RadialDistortionImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	size_t in_h = *height;
	size_t in_w = *width;
	uchar4* d_in, * d_out;
	float scale;
	if (k1 > 0) scale = 1 - k1 / 2;
	else scale = 1 - k1 / 4;
	hipMalloc(&d_in, in_h * in_w * sizeof(uchar4));
	hipMalloc(&d_out, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_in, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	radial <<< dim3(1 + ((in_w - 1) / 32), 1 + ((in_h - 1) / 32), 1), dim3(32, 32, 1) >>> (d_in, d_out, in_h, in_w, k1, scale);
	hipMemcpy(*image, d_out, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
}

std::string RadialDistortionImageCommand::toString() {
	return "RadialDistortion(" + std::to_string(k1) + ")";
}

__global__ void contrast(uchar4* image, size_t height, size_t width, float alpha) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;
		image[idx].x = alpha * image[idx].x;
		image[idx].y = alpha * image[idx].y;
		image[idx].z = alpha * image[idx].z;
		if (image[idx].x > 255)
			image[idx].x = 255;
		if (image[idx].y > 255)
			image[idx].y = 255;
		if (image[idx].z > 255)
			image[idx].z = 255;
	}
}

void executeContrast(uchar4** image, size_t* height, size_t* width, float alpha) {
	size_t in_h = *height;
	size_t in_w = *width;
	uchar4* d_image;
	hipMalloc(&d_image, in_h * in_w * sizeof(uchar4));
	hipMemcpy(d_image, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	contrast <<< dim3(1 + ((in_h - 1) / 32), 1 + ((in_w - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, in_h, in_w, alpha);
	hipMemcpy(*image, d_image, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
}

__constant__ int mask[3 * 3];

__global__ void sharpen(uchar4* image_in, uchar4* image_out, size_t height, size_t width) {
	extern __shared__ uchar4 sh[];
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < width && y < height) {
		unsigned int sh_w = blockDim.x + 2;
		unsigned int sh_h = blockDim.y + 2;

		unsigned int sh_x = threadIdx.x + 1;
		unsigned int sh_y = threadIdx.y + 1;

		unsigned int idx = y * width + x;

		sh[sh_w * sh_y + sh_x] = image_in[idx];
		uchar4 padding = image_in[idx]; //same padding // pos : sh_w*sh_y+sh_x
		if (sh_x == 1) {
			sh[sh_w * sh_y] = padding;
			if(sh_y == 1) sh[sh_w] = padding;
			else if (sh_y == sh_h-2 || y == height - 1) sh[sh_w * (sh_y + 1)] = padding;
		}
		else if (sh_x == sh_w-2 || x == width - 1) {
			sh[sh_w * sh_y + (sh_x + 1)] = padding;
			if (sh_y == 1) sh[sh_x + 1] = padding;
			else if (sh_y == sh_h - 2 || y == height - 1) sh[sh_w * (sh_y + 1) + (sh_x + 1)] = padding;
		}
		if (sh_y == 1) sh[sh_x] = padding;
		else if (sh_y == sh_h-2 || y == height - 1) sh[sh_w * (sh_y + 1) + sh_x] = padding;
		__syncthreads();

		int tempx = 0;
		int tempy = 0;
		int tempz = 0;
		for (int j = 0; j < 3; j++) {
			for (int i = 0; i < 3; i++) {
				unsigned int sh_pos = (sh_y - 1 + j) * sh_w + (sh_x - 1 + i);

				tempx += mask[j * 3 + i] * sh[sh_pos].x;
				tempy += mask[j * 3 + i] * sh[sh_pos].y;
				tempz += mask[j * 3 + i] * sh[sh_pos].z;
			}
		}
		tempx = tempx > 255 ? 255 : (tempx < 0 ? 0 : tempx);
		tempy = tempy > 255 ? 255 : (tempy < 0 ? 0 : tempy);
		tempz = tempz > 255 ? 255 : (tempz < 0 ? 0 : tempz);
		image_out[idx].x = tempx;
		image_out[idx].y = tempy;
		image_out[idx].z = tempz;
	}
}

void SharpeningImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
	size_t in_h = *height;
	size_t in_w = *width;
	uchar4 *d_in, *d_out;
	int filter[] = { 0,-1,0,-1,5,-1,0,-1,0 };
	hipMalloc(&d_in, in_h * in_w * sizeof(uchar4));
	hipMalloc(&d_out, in_h * in_w * sizeof(uchar4));
	hipMemcpyToSymbol(HIP_SYMBOL(mask), filter, 3*3*sizeof(int));
	hipMemcpy(d_in, *image, in_h * in_w * sizeof(uchar4), hipMemcpyHostToDevice);
	sharpen <<< dim3(1+((in_w-1)/32), 1+((in_h-1)/32), 1), dim3(32, 32, 1), (32+2)*(32+2)*sizeof(uchar4)>>> (d_in, d_out, in_h, in_w);
	hipMemcpy(*image, d_out, in_h * in_w * sizeof(uchar4), hipMemcpyDeviceToHost);
}

std::string SharpeningImageCommand::toString() {
	return "Sharpen()";
}


__global__ void rgba_to_xyYA_separated(
	uchar4* rgba_in,
	float* x_out, float* y_out, float* logY_out, float* A_out,
	size_t height, size_t width,
	float delta
) {
	int in_x = blockIdx.x * blockDim.x + threadIdx.x;
	int in_y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = in_y * width + in_x;
	if (in_x < width && in_y < height) {
		float r = rgba_in[idx].x / 255.f;
		float g = rgba_in[idx].y / 255.f;
		float b = rgba_in[idx].z / 255.f;
		float a = rgba_in[idx].w / 255.f;

		float X = (r * 0.4124f) + (g * 0.3576f) + (b * 0.1805f);
		float Y = (r * 0.2126f) + (g * 0.7152f) + (b * 0.0722f);
		float Z = (r * 0.0193f) + (g * 0.1192f) + (b * 0.9505f);

		float L = X + Y + Z;
		float x = X / L;
		float y = Y / L;

		float logY = log10f(delta + Y);

		x_out[idx] = x;
		y_out[idx] = y;
		logY_out[idx] = logY;
		A_out[idx] = a;
	}
}

__global__ void min_max(float* arr, size_t length, float* minval, float* maxval) {
	extern __shared__ float2 sh[];
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;
	sh[tid].x = arr[myId];
	sh[tid].y = arr[myId];
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sh[tid].x = __min(sh[tid].x, sh[tid + s].x);
			sh[tid].y = __max(sh[tid].y, sh[tid + s].y);
		}
		__syncthreads();
	}
	if(tid==0) atomicAdd
}


//
//void ToneMappingImageCommand::execute(uchar4** image, size_t* height, size_t* width) {
//	size_t in_h = *height;
//	size_t in_w = *width;
//	size_t numPixels = in_h * in_w;
//	dim3 gridSize(1+((in_w-1)/32), 1+((in_h-1)/32), 1);
//	dim3 blockSize(32, 32, 1);
//	uchar4* d_in;
//	float *d_x, *d_y, *d_logY, *d_A;
//	hipMalloc(&d_x, numPixels * sizeof(float));
//	hipMalloc(&d_y, numPixels * sizeof(float));
//	hipMalloc(&d_logY, numPixels * sizeof(float));
//	hipMalloc(&d_A, numPixels * sizeof(float));
//	hipMalloc(&d_in, numPixels * sizeof(uchar4));
//	hipMemcpy(d_in, *image, numPixels * sizeof(uchar4), hipMemcpyHostToDevice);
//	rgba_to_xyYA_separated<<<gridSize, blockSize>>>(d_in, d_x, d_y, d_logY, d_A, in_h, in_w, 0.0001f);
//	const int numBins = 1024;
//	unsigned int* d_cdf;
//	hipMalloc(&d_cdf, sizeof(unsigned int) * numBins);
//
//	//TODO
//	/*Here are the steps you need to implement
//	  1) find the minimum and maximum value in the input logLuminance channel
//		 store in min_logLum and max_logLum
//	  2) subtract them to find the range
//	  3) generate a histogram of all the values in the logLuminance channel using
//		 the formula: bin = (lum[i] - lumMin) / lumRange * numBins
//	  4) Perform an exclusive scan (prefix sum) on the histogram to get
//		 the cumulative distribution of luminance values (this should go in the
//		 incoming d_cdf pointer which already has been allocated for you)       */
//	float *d_minlogY, * d_maxlogY;
//	hipMalloc(&d_minlogY, sizeof(float));
//	hipMalloc(&d_maxlogY, sizeof(float));
//	min_max <<<1+(numPixels-1)/1024, 1024, 1024*size(float2) >>> (d_logY, numPixels, minlogY, maxlogY);
//	float h_minlogY, h_maxlogY;
//	hipMemcpy(&h_minlogY, d_minlogY, sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(&h_maxlogY, d_maxlogY, sizeof(float), hipMemcpyDeviceToHost);
//}
//
//
//
//std::string ToneMappingImageCommand::toString() {
//	return "ToneMap()";
//}